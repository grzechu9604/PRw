#include "hip/hip_runtime.h"
/*
grid wieloblokowy, jeden w�tek oblicza 2 lub 4 (podzia� pracy dwuwymiarowy)
s�siednich element�w macierzy wynikowej, obliczenia przy wykorzystaniu pami�ci wsp�dzielonej bloku w�tk�w,
*/
/**
* Matrix multiplication: C = A * B.
* Host code.jk
*
* This sample implements matrix multiplication which makes use of shared memory
* to ensure data reuse, the matrix multiplication is done using tiling approach.
* It has been written for clarity of exposition to illustrate various CUDA programming
* principles, not with the goal of providing the most performant generic kernel for matrix multiplication.

*/

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <conio.h>

template <int BLOCK_SIZE> __global__ void MatrixMulKernel_5(float *A, float *B, float *C, int WIDTH) {
		// Block index
		int bx = blockIdx.x;
		int by = blockIdx.y;

		// Thread index
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		// Index of the first sub-matrix of A processed by the block
		int aBegin = WIDTH * BLOCK_SIZE * by;

		// Index of the last sub-matrix of A processed by the block
		int aEnd = aBegin + WIDTH - 1;

		// Step size used to iterate through the sub-matrices of A
		int aStep = BLOCK_SIZE;

		// Index of the first sub-matrix of B processed by the block
		int bBegin = BLOCK_SIZE * bx;

		// Step size used to iterate through the sub-matrices of B
		int bStep = BLOCK_SIZE * WIDTH;

		// Csub is used to store the element of the block sub-matrix
		// that is computed by the thread
		float Csub = 0;

		// Loop over all the sub-matrices of A and B
		// required to compute the block sub-matrix
		for (int a = aBegin, b = bBegin;
			a <= aEnd;
			a += aStep, b += bStep) {
			// Declaration of the shared memory array As used to
			// store the sub-matrix of A
			__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

			// Declaration of the shared memory array Bs used to
			// store the sub-matrix of B
			__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

			// Load the matrices from device memory
			// to shared memory; each thread loads
			// one element of each matrix
			As[ty][tx] = A[a + WIDTH * ty + tx];
			Bs[ty][tx] = B[b + WIDTH * ty + tx];

			// Synchronize to make sure the matrices are loaded
			__syncthreads();

			// Multiply the two matrices together;
			// each thread computes one element
			// of the block sub-matrix
#pragma unroll

			for (int k = 0; k < BLOCK_SIZE; ++k) {
				Csub += As[ty][k] * Bs[k][tx];
			}

			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of A and B in the next iteration
			__syncthreads();
		}

		// Write the block sub-matrix to device memory;
		// each thread writes one element
		int c = WIDTH * BLOCK_SIZE * by + BLOCK_SIZE * bx;
		C[c + WIDTH * ty + tx] = Csub;
}

void ConstantInit(float *data, int size, float val) {
	for (int i = 0; i < size; ++i) {
		data[i] = val;
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int MatrixMultiply(int block_size, const dim3 &dimsA, const dim3 &dimsB) {
	// Allocate host memory for matrices A and B
	unsigned int size_A = dimsA.x * dimsA.y;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
	unsigned int size_B = dimsB.x * dimsB.y;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

	// Initialize host memory
	const float valB = 0.01f;
	ConstantInit(h_A, size_A, 1.0f);
	ConstantInit(h_B, size_B, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;

	// Allocate host matrix C
	dim3 dimsC(dimsA.x, dimsA.y, 1);
	unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

	if (h_C == NULL) {
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}


	ConstantInit(h_C, dimsC.x * dimsC.y, 1.0f);

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

	// Setup execution parameters
	dim3 threads(block_size, 1);
	dim3 grid(1, 1); //DLACZEGO (1,1)?

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");
	// Performs warmup operation using matrixMul CUDA kernel
	MatrixMulKernel_5 <16><<<grid,threads>>>(d_A, d_B, d_C, dimsA.x);
	printf("done\n");
	hipDeviceSynchronize();

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	checkCudaErrors(hipEventCreate(&start));
	hipEvent_t stop;
	checkCudaErrors(hipEventCreate(&stop));

	// Record the start event
	checkCudaErrors(hipEventRecord(start, NULL));
	// Execute the kernel
	int nIter = 300;
	for (int j = 0; j < nIter; j++) {
		MatrixMulKernel_5 <16><<<grid,threads>>>(d_A, d_B, d_C, dimsA.x);
	}
	// Record the stop event
	checkCudaErrors(hipEventRecord(stop, NULL));
	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
		static_cast<double>(dimsA.y) *
		static_cast<double>(dimsB.x);
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
		(msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s\n Time= %.3f msec\n Size= %.0f Ops\n" \
		" WorkgroupSize= %u threads/block\n\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);

	// Copy result from device to host
	checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
	printf("Checking computed result for correctness: ");
	bool correct = true;
	// test relative error by the formula
	//     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
	double eps = 1.e-2;  // machine zero
	for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
		double abs_err = fabs(h_C[i] - (dimsA.x * valB));
		double dot_length = dimsA.x;
		double abs_val = fabs(h_C[i]);
		double rel_err = abs_err / abs_val / dot_length;

		if (rel_err > eps) {
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x * valB, eps);
			correct = false;
		}
	}
	printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_B));
	checkCudaErrors(hipFree(d_C));

	printf("\nNOTE: The CUDA Samples are not meant for performance"\
		"measurements. Results may vary when GPU Boost is enabled.\n");

	if (correct) {
		return EXIT_SUCCESS;
	}
	else {
		return EXIT_FAILURE;
	}
}


/**
* Program main
*/
int main(int argc, char **argv) {
	printf("[Matrix Multiply Using CUDA] - Starting...\n");

	// This will pick the best possible CUDA capable device, otherwise
	// override the device ID based on input provided at the command line
	int dev = findCudaDevice(argc, (const char **)argv);

	int block_size = 16;
	dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
	dim3 dimsB(5 * 2 * block_size, 5 * 2 * block_size, 1);

	if (dimsA.x != dimsB.y) {
		printf("Error: outer matrix dimensions must be equal. (%d != %d)\n", dimsA.x, dimsB.y);
		exit(EXIT_FAILURE);
	}
	printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

	int matrix_result = MatrixMultiply(block_size, dimsA, dimsB);

	printf("End of program [matrix_result = %d]\n", matrix_result);
	getch();
	exit(matrix_result);
}